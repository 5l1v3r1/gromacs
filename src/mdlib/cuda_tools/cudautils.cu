#include <stdlib.h>

#include "gmx_fatal.h"
#include "smalloc.h"

#include "hip/hip_runtime.h"
#include "cudautils.h"

/*** General CUDA data operations ***/
/* TODO: create a cusmalloc module that implements similar things as smalloc */

static int cu_copy_D2H_generic(void * h_dest, void * d_src, size_t bytes, 
                               gmx_bool async = FALSE, hipStream_t stream = 0)
{
    hipError_t stat;
    
    if (h_dest == 0 || d_src == 0 || bytes <= 0)
        return -1;

    if (async)
    {
        stat = hipMemcpyAsync(h_dest, d_src, bytes, hipMemcpyDeviceToHost, stream);
        CU_RET_ERR(stat, "DtoH hipMemcpyAsync failed");

    }
    else
    {
        stat = hipMemcpy(h_dest, d_src, bytes, hipMemcpyDeviceToHost);
        CU_RET_ERR(stat, "DtoH hipMemcpy failed");
    }

    return 0;
}

int cu_copy_D2H(void * h_dest, void * d_src, size_t bytes)
{
    return cu_copy_D2H_generic(h_dest, d_src, bytes, FALSE);
}

int cu_copy_D2H_async(void * h_dest, void * d_src, size_t bytes, hipStream_t stream = 0)
{
    return cu_copy_D2H_generic(h_dest, d_src, bytes, TRUE, stream);
}

int cu_copy_D2H_alloc(void ** h_dest, void * d_src, size_t bytes)
{ 
    if (h_dest == 0 || d_src == 0 || bytes <= 0)
        return -1;

    smalloc(*h_dest, bytes);

    return cu_copy_D2H(*h_dest, d_src, bytes);
}


static int cu_copy_H2D_generic(void * d_dest, void * h_src, size_t bytes, 
                               gmx_bool async = FALSE, hipStream_t stream = 0)
{
    hipError_t stat;

    if (d_dest == 0 || h_src == 0 || bytes <= 0)
        return -1;

    if (async)
    {
        stat = hipMemcpyAsync(d_dest, h_src, bytes, hipMemcpyHostToDevice, stream);
        CU_RET_ERR(stat, "HtoD hipMemcpyAsync failed");
    }
    else
    {
        stat = hipMemcpy(d_dest, h_src, bytes, hipMemcpyHostToDevice);
        CU_RET_ERR(stat, "HtoD hipMemcpy failed");
    }

    return 0;
}

int cu_copy_H2D(void * d_dest, void * h_src, size_t bytes)
{   
    return cu_copy_H2D_generic(d_dest, h_src, bytes, FALSE);
}

int cu_copy_H2D_async(void * d_dest, void * h_src, size_t bytes, hipStream_t stream = 0)
{   
    return cu_copy_H2D_generic(d_dest, h_src, bytes, TRUE, stream);
}

int cu_copy_H2D_alloc(void ** d_dest, void * h_src, size_t bytes)
{
    hipError_t stat;

    if (d_dest == 0 || h_src == 0 || bytes <= 0)
        return -1;

    stat = hipMalloc(d_dest, bytes);
    CU_RET_ERR(stat, "hipMalloc failed in cu_copy_H2D_alloc");

    return cu_copy_H2D(*d_dest, h_src, bytes);
}

int cu_wait_event(hipEvent_t stop, hipEvent_t start, float *time)
{
    hipError_t s;

    s = hipEventSynchronize(stop);
    CU_RET_ERR(s, "hipEventSynchronize failed in cu_wait_event");

    s = hipEventElapsedTime(time, start, stop);
    CU_RET_ERR(s, "hipEventElapsedTime failed in cu_wait_event");

    return 0;
}

/* Binds texture with name tex_name to the GPU global memory (of size elements) 
   pointed by d_ptr.
   Returns the offset that needs to be used when fetching from the texture.
 */
template <typename T>
size_t cu_bind_texture(const char *tex_name, const T *d_ptr, int size)
{
    hipError_t             stat;
    hipChannelFormatDesc   cd;
    const textureReference  *tex;
    char                    str[100];

    size_t offset;

    stat = hipGetTextureReference(&tex, HIP_SYMBOL(tex_name));
    sprintf(str, "hipGetTextureReference on %s failed", tex_name);
    CU_RET_ERR(stat, str);
    cd = hipCreateChannelDesc<T>();

    stat = hipBindTexture(&offset, tex, d_ptr, &cd, size*sizeof(*d_ptr));
    sprintf(str, "hipBindTexture on %s failed ", tex_name);
    CU_RET_ERR(stat, str);

    return offset;
}

/* Instantiate cu_bind_texture with float */
template size_t cu_bind_texture<float>(const char *, const float *, int);

/*! Unbinds texture with name tex_name. */
void cu_unbind_texture(const char *tex_name)
{
    hipError_t             stat;
    const textureReference  *tex;
    char                    str[100];

    stat = hipGetTextureReference(&tex, HIP_SYMBOL(tex_name));
    sprintf(str, "hipGetTextureReference on %s failed", tex_name);
    CU_RET_ERR(stat, str);
    stat = hipUnbindTexture(tex);
    sprintf(str, "hipUnbindTexture on %s failed ", tex_name);
    CU_RET_ERR(stat, str);
}

/*! Caculates and returns the time difference between event start and stop. */
float cu_event_elapsed(hipEvent_t start, hipEvent_t stop)
{
    float t = 0.0;
    hipError_t stat;

    stat = hipEventElapsedTime(&t, start, stop);
    CU_RET_ERR(stat, "hipEventElapsedTime failed in cu_event_elapsed");

    return t;
}


/**** Operation on buffered arrays (arrays with "over-allocation" in gmx wording) */
/*! Frees the device memory pointed by d_ptr and resets the associated 
 *  size and allocation size variables to -1.
 */
void cu_free_buffered(void *d_ptr, int *n, int *nalloc)
{
    hipError_t stat;

    if (d_ptr)
    {
        stat = hipFree(d_ptr);
        CU_RET_ERR(stat, "hipFree failed");
    }

    if (n)
    {
        *n = -1;
    }

    if (nalloc)
    {
        *nalloc = -1;
    }
}

/*! Reallocates the device memory pointed by d_ptr and copies the data from the 
 * location pointed by h_src host-side pointer. Allocation is buffered and 
 * therefor freeing is only needed if the previously allocated space is not 
 * enough. 
 */
void cu_realloc_buffered(void **d_dest, void *h_src, size_t type_size,
                                    int *curr_size, int *curr_alloc_size,
                                    int req_size,
                                    hipStream_t stream,
                                    gmx_bool doAsync)
{
    hipError_t stat;

    if (d_dest == NULL || req_size < 0)
    {
        return;
    }

    /* reallocate only if the data does not fit = allocation size is smaller 
       than the current requested size */
    if (req_size > *curr_alloc_size)
    {
        /* only free if the array has already been initialized */
        if (*curr_alloc_size >= 0)
        {
            cu_free_buffered(*d_dest, curr_size, curr_alloc_size);
        }

        *curr_alloc_size = 1.2 * req_size + 100;

        stat = hipMalloc(d_dest, *curr_alloc_size * type_size);
        CU_RET_ERR(stat, "hipMalloc failed in cu_free_buffered");
    }

    /* size could have changed without actual reallocation */
    *curr_size = req_size;

    /* upload to device */
    if (h_src)
    {
        if (doAsync)
        {
            cu_copy_H2D_async(*d_dest, h_src, *curr_size * type_size, stream);
        }
        else
        {
            cu_copy_H2D(*d_dest, h_src,  *curr_size * type_size);
        }
    }
}
